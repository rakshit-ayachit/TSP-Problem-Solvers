#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include ""

#define V 10

__global__ void tsp(int* graph, int n, int* dp) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= (1 << n)) return;

    for (int j = 0; j < n; j++) {
        if (i & (1 << j)) {
            int prev_i = i & ~(1 << j);
            int min_cost = INT_MAX;
            for (int k = 0; k < n; k++) {
                if (graph[k * n + j] != INT_MAX && (i == (1 << j) || (prev_i & (1 << k)))) {
                    int cost = dp[prev_i * n + k] + graph[k * n + j];
                    if (cost < min_cost) min_cost = cost;
                }
            }
            if (min_cost != INT_MAX) {
                dp[i * n + j] = min_cost;
            }
        }
    }
}

int main() {
    int graph[V][V];
    srand(time(NULL));
    for (int i = 0; i < V; i++) {
        for (int j = 0; j < V; j++) {
            if (i == j)
                graph[i][j] = 0;
            else
                graph[i][j] = rand() % 100;
        }
    }
    
    int* d_graph;
    hipMalloc((void**)&d_graph, V * V * sizeof(int));
    hipMemcpy(d_graph, graph, V * V * sizeof(int), hipMemcpyHostToDevice);

    int dp_size = (1 << V) * V * sizeof(int);
    int* d_dp;
    hipMalloc((void**)&d_dp, dp_size);
    hipMemset(d_dp, 0x7F, dp_size);

    for (int i = 0; i < V; i++) {
        int start_state = (1 << i) * V + i;
        hipMemset(&d_dp[start_state], 0, sizeof(int));
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    tsp << <(1 << V) / 256 + 1, 256 >> > (d_graph, V, d_dp);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    int* dp = (int*)malloc(dp_size);
    hipMemcpy(dp, d_dp, dp_size, hipMemcpyDeviceToHost);

    int min_route_cost = INT_MAX;
    int final_mask = (1 << V) - 1;
    for (int i = 0; i < V; i++) {
        if (dp[final_mask * V + i] < min_route_cost) {
            min_route_cost = dp[final_mask * V + i];
        }
    }
    
	 printf("Random input matrix:\n");
    for (int i = 0; i < V; i++) {
        for (int j = 0; j < V; j++) {
            printf("%d\t", graph[i][j]);
        }
        printf("\n");
    }
    printf("\n");
	printf("This is the Parallelized Approach with CUDA: \n");
    printf("Minimum cost: %d\n", min_route_cost);
    printf("Time taken: %f seconds\n", milliseconds / 1000);

    hipFree(d_graph);
    hipFree(d_dp);
    free(dp);

    return 0;
}

